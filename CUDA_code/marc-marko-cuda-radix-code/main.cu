#include <iostream>
#include <algorithm>
#include <iomanip>
#include <ctime>
#include "kernel.cuh"
#include<chrono>

void cpu_sort(unsigned int* h_out, unsigned int* h_in, size_t len)
{
    for (int i = 0; i < len; ++i)
    {
        h_out[i] = h_in[i];
    }
    std::sort(h_out, h_out + len);
}

int main()
{
    for (int shift_size = 16; shift_size < 30; shift_size++)
    {
    
        unsigned int num_elems = (1 << shift_size);
        unsigned int* h_in = new unsigned int[num_elems];
        unsigned int* h_in_rand = new unsigned int[num_elems];
        unsigned int* h_out_gpu = new unsigned int[num_elems];
        unsigned int* h_out_cub = new unsigned int[num_elems];
        unsigned int* h_out_cpu = new unsigned int[num_elems];
        for (int j = 0; j < num_elems; j++)
        {
            h_in[j] = (num_elems - 1) - j;
            h_in_rand[j] = rand() % num_elems;
        }
        auto start_cpu = std::chrono::high_resolution_clock::now();
        cpu_sort(h_out_cpu, h_in_rand, num_elems);  
        auto elapsed_cpu = std::chrono::high_resolution_clock::now() - start_cpu;
        long long microseconds_cpu = std::chrono::duration_cast<std::chrono::microseconds>(elapsed_cpu).count();
        std::cout <<"CPU radix sort in µs: "<<  microseconds_cpu << std::endl;
        

        unsigned int* d_in;
        unsigned int* d_out;
        unsigned int* d_out_cub;
        hipMalloc(&d_in, sizeof(unsigned int) * num_elems);
        hipMalloc(&d_out, sizeof(unsigned int) * num_elems);
        hipMalloc(&d_out_cub, sizeof(unsigned int) * num_elems);
        hipMemcpy(d_in, h_in_rand, sizeof(unsigned int) * num_elems, hipMemcpyHostToDevice);

        auto start = std::chrono::high_resolution_clock::now();
        radix_sort(d_out, d_in, num_elems);
        auto elapsed = std::chrono::high_resolution_clock::now() - start;
        long long microseconds = std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
        std::cout <<"Own implementation in µs: " << microseconds << std::endl;
        
        hipMemcpy(h_out_gpu, d_out, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost);

        
        void     *d_temp_storage = NULL;
        size_t   temp_storage_bytes = 0;
        hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out_cub, num_elems);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run sorting operation
        auto start_cub = std::chrono::high_resolution_clock::now();
        hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out_cub, num_elems);
        auto elapsed_cub = std::chrono::high_resolution_clock::now() - start;
        long long microseconds_cub = std::chrono::duration_cast<std::chrono::microseconds>(elapsed_cub).count();
        std::cout <<"CUB radix sort in µs: "<<  microseconds_cub << std::endl;

        hipMemcpy(h_out_cub, d_out_cub, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost);

        bool match = true;


        for (int i = 0; i < num_elems; ++i)
        {
            if (h_out_cpu[i] != h_out_gpu[i])
            {
                match = false;
            }
        }
        std::cout << std::boolalpha;   
        std::cout << "Match: " << match << std::endl;
        std::cout <<std::endl;

        hipFree(d_out);
        hipFree(d_in);
        hipFree(d_out_cub);
        free(h_in);
        free(h_in_rand);
        free(h_out_gpu);
        free(h_out_cub);
        free(h_out_cpu);
        hipFree(d_temp_storage);
    }
}