#include "hip/hip_runtime.h"
#include<iostream>
#include<chrono>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<string>
#include<hipcub/hipcub.hpp>
#include"kernel.cuh"
#include"myKernel.cuh"

#define ARRAY_SIZE 1e2
#define BLOCK_SIZE 256
#define SEED 42

typedef unsigned int datatype;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        //fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line); 
        if (abort) exit(code);
    }
}


__global__ void init_arr(datatype* data, unsigned long seed, int array_length){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < array_length){
      hiprandState state;
      hiprand_init(seed, idx, 0, &state);
      data[idx] = (datatype)hiprand(&state) % 10000;
    }
}


void print_cuda_array(datatype* array, size_t array_length){
  datatype* cpu_data = (datatype*)malloc(array_length*sizeof(datatype));
  hipMemcpy(cpu_data, array,array_length*sizeof(datatype), hipMemcpyDeviceToHost);

  std::cout << "[";
  for(size_t i = 0; i < array_length; i++) {
    std::string str = std::to_string(cpu_data[i]);
    std::cout << str;
    if( i < array_length - 1) std::cout << ", ";
  }
  std::cout << "]\n";
}


int main() {
  //Init data
  const size_t N = ARRAY_SIZE;

  datatype *data_in;
  datatype *data_out;
  datatype *data_out_cub;

  hipMalloc(&data_in, N * sizeof(datatype));
  hipMalloc(&data_out, N * sizeof(datatype));
  hipMalloc(&data_out_cub, N * sizeof(datatype));
  
  init_arr<<<ARRAY_SIZE / BLOCK_SIZE + 1,BLOCK_SIZE>>>(data_in, SEED, N);
  //print_cuda_array(data_in, N);

  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, data_in, data_out_cub, N);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipDeviceSynchronize();
  hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, data_in, data_out_cub, N);
  hipDeviceSynchronize();
  std::cout << "Post Process cub\n";
  std::cout << "Input Array\n";
  //print_cuda_array(data_in, N);
  std::cout << "Output Array\n";
  //print_cuda_array(data_out_cub, N);

  Kernels::radix_sort(data_in, data_out, N);

  radix_sort((unsigned int*)data_out, (unsigned int*)data_in, N);
  std::cout << "Post Process M&M's\n";
  std::cout << "Input Array\n";
  //print_cuda_array(data_in, N);
  std::cout << "Output Array\n";
  //print_cuda_array(data_out, N);
  
  hipFree(data_in); 
  hipFree(data_out);
  hipFree(data_out_cub);

  return 0;
}